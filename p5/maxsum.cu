/**
  * @file maxsum.cu
  * @author Jake Donovan (jmpatte8)
  * This file utilizes CUDA functions in order to calculate the maximum sum of a number of values by utilizing threads and CUDA functions.
*/

// Elapsed Real Time for input-5.txt: (real) = 1.172 seconds.
// Type of GPU: RTX 2070

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Input sequence of values.
int * vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of values.
int vCap = 0;

// Our results array which holds all local max sums calculated by each thread
int * results;


/**
  * General function to report a failure and exit.
  * @param message the passed error message
*/
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: maxsum [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

/**
  * This function is responsible for calculating a local max value starting at an indicated position and saves to our output
  * array using the values from the passed input_vList to determine the localMax starting from the indicated position, vCount is the total number
  * of values in our vList, and report is a boolean that lets us know whether each thread should report their localMax or simply save the localMax to our results
  * array
  * @param input_vList our passed values that we want to use to determing our local max for each thread
  * @param output our results array which holds each localMax calculated by our threads
  * @param vCount the number of values in our vList array aka input_vList
  * @param report a boolean that alerts our threads whether they should report their localMax or only save their value to our results array ( aka the output param )
*/
__global__ void checkSum( int * input_vList, int * output, int vCount, bool report ) {
  // Compute a unique index for this thread, based on its location in its block location in its grid
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  // make sure our thread idx is not larger than the number of values we should examine
  // this helps in the case where we have more threads than we need to calculate our max sum
  if ( idx < vCount ) { 
    // our current sum calculated by current thread
    int currentSum = 0;
    // max sum found by current thread
    int localMax = 0;

    // go through each value in input list starting at idx to find the local max
    for( int i = idx; i >= 0; i-- ) {
      // update current sum
      currentSum += input_vList[ i ];

      // is our current sum larger than our current local max?
      // then update it
      if( currentSum > localMax ) {
        localMax = currentSum;
      }
    }

    // update our output list
    output[ idx ] = localMax;

    // if report is true then print our thread at idx and our localMax found by our current thread
    if( report ) {
      printf( "I'm thread %d. The maximum sum I found is %d\n", idx, localMax );
    }
  }
}

/**
  * Use command line arguments, CUDA functions, and this file's functions in order to determine the max sum of a list of values
  * and report the max sum found
  * @param argc the number of command line arguments
  * @param argv pointers to each command line argument (as strings)
  * @return program exit status
*/
int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();

  // get space for results array
  // vCount = cap for results
  results = ( int * )malloc( vCount * sizeof( int ) );
  
  // Add code to allocate memory on the device and copy over the list.
  int *devList = NULL;

  if( hipMalloc( ( void **)&devList, vCount * sizeof( int ) ) != hipSuccess ) {
    fail( "Failed to allocate space for length list on device" );
  }

  // Add code to copy the list over to the device.
  if( hipMemcpy( devList, vList, vCount * sizeof( int ), hipMemcpyHostToDevice ) != hipSuccess ) {
    fail( "Failed to copy list to device" );
  }

  // create device pointer for our output array
  int *devResult = NULL;

  // Add code to allocate space on the device to hold the results.
  if( hipMalloc( ( void ** )&devResult, vCount * sizeof( int ) ) != hipSuccess ) {
    fail( "Failed to allocate space for results list on device" );
  }

  // Block and grid dimensions. ( HOW MANY THREADS I NEED FOR GPU TASK )
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions (you'll need to add some parameters)
  checkSum<<<blocksPerGrid, threadsPerBlock>>>( devList, devResult, vCount, report );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Add code to copy results back to the host, compare the local largest products
  // and report the final largest product
  if( hipMemcpy( results, devResult, vCount * sizeof( int ), hipMemcpyDeviceToHost ) != hipSuccess ) {
    fail( "Can't copy list from device to host" );
  }

  // save our largest max
  int maxSum = 0;

  // report final largest sum
  for( int i = 0; i < vCount; i++ ) {
    // if current localMax in results array is larger than maxSum, then update maxSum
    if( results[ i ] > maxSum ) {
      maxSum = results[ i ];
    }
  }

  // report maxSum found
  printf( "Maximum Sum: %d\n", maxSum );

  // Free memory on the device and the host.
  hipFree( devList );
  // free vList
  free( vList );
  // free results array
  free( results );

  // reset
  hipDeviceReset();
  // return successfully
  return 0;
}
